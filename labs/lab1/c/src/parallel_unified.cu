#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "reader.h"
#include <stdlib.h>
#include "gputimer.h"

__global__ void kernel(int* data, int* result, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		if (*(data+idx*3+2)==0)
		{
			//and
			*(result+idx) = *(data+idx*3) && *(data+idx*3+1);
		} 
		else if (*(data+idx*3+2)==1)
		{
			//or
			*(result+idx) = *(data+idx*3) || *(data+idx*3+1);
		}
		else if (*(data+idx*3+2)==2)
		{
			//nand
			*(result+idx) = !(*(data+idx*3) && *(data+idx*3+1));
		}
		else if (*(data+idx*3+2)==3)
		{
			//nor
			*(result+idx) = !(*(data+idx*3) || *(data+idx*3+1));
		}
		else if (*(data+idx*3+2)==4)
		{
			//xor
			*(result+idx) = *(data+idx*3) ^ *(data+idx*3+1);
		}
		else if (*(data+idx*3+2)==5)
		{
			//xnor
			*(result+idx) = !(*(data+idx*3) ^ *(data+idx*3+1));
		}
		else
		{
			//invalid
			*(result+idx) = -1;
		}
		//*(result+idx)=idx;
	}
}

int main(int argc, char *argv[])
{
	GpuTimer timer;
	timer.Start();
	printf("ECSE 420 Lab 1: Logic Gates Simulation - parallel_unified\n");
	if (argc != 4)
	{
		printf("RuntimeError: Wrong inputs.\n");
		printf("You should input follows this format: ./parallel_unified <input_file_path> <input_file_length> <output_file_path>\n");
		return 1;
	}
	
	int size = atoi(argv[2]);

	GpuTimer loadTimer;
	loadTimer.Start();
	int* cuda_file;
	hipMallocManaged((void **) &cuda_file, size*3*sizeof(int));

	if(read_csv_array(argv[1], cuda_file, &size)==0){
		
		int *cuda_output;
		hipMallocManaged((void **) &cuda_output, size*sizeof(int));
		loadTimer.Stop();
		float loadTime = loadTimer.Elapsed();

		int block = size/1024;
		if (size%1024) {
			block++;
		}
		
		GpuTimer parallelTimer;
		parallelTimer.Start();
		kernel <<<block, 1024>>> (cuda_file, cuda_output, size);
		hipDeviceSynchronize();

		parallelTimer.Stop();

		float parallelTime = parallelTimer.Elapsed();
		
		GpuTimer retriveTimer;
		retriveTimer.Start();
		save(argv[3], cuda_output, size);
		retriveTimer.Stop();
		float retriveTime = retriveTimer.Elapsed();

		hipFree(cuda_file);
		hipFree(cuda_output);
		
		timer.Stop();
		float totalTime = timer.Elapsed();
		
		printf("Done\n");
		printf("Load Time: %f ms\nParallel Time: %f ms\nRetrive Time: %f ms\n---------------\nTotal Time: %f ms\n", loadTime, parallelTime, retriveTime, totalTime);

		return 0;
	}
	else
	{
		timer.Stop();
		hipFree(cuda_file);
		return 1;
	}
	
}
