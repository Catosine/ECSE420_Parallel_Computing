#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define GRID_SIZE 4
#define MIU 0.0002
#define RHO 0.5
#define G 0.75

__global__ void simulation_kernel(float *grid, float *grid_1, float *grid_2)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int y = idx/GRID_SIZE;
    int x = idx%GRID_SIZE;
    if(y==0&&x!=0&&x!=GRID_SIZE-1)
    {//boundary condition 1: y = 0
        float u1_x_1 = *(grid_1+GRID_SIZE*(y+1)+x);
        *(grid+GRID_SIZE*y+x) = G * u1_x_1;
    } 
    else if (y==GRID_SIZE-1&&x!=0&&x!=GRID_SIZE-1)
    {//boundary condition 2: y = GRID_SIZE-1
        float u1_x_y1 = *(grid_1+GRID_SIZE*(y-1)+x);
        *(grid+GRID_SIZE*y+x) = G * u1_x_y1;
    }
    else if(x==0&&y!=0&&y!=GRID_SIZE-1)
    {//boundary condition 3: x = 0
        float u1_1_y = *(grid_1+GRID_SIZE*y+(x+1));
        *(grid+GRID_SIZE*y+x) = G * u1_1_y;
    } 
    else if (x==GRID_SIZE-1&&y!=0&&y!=GRID_SIZE-1)
    {//boundary condition 4: x = GRID_SIZE-1
        float u1_x1_y = *(grid_1+GRID_SIZE*y+(x-1));
        *(grid+GRID_SIZE*y+x) = G * u1_x1_y;
    }
    else if (x==0&&y==0) 
    {// corner condition 1: x = y = 0
        float u1_1_0 = *(grid_1+GRID_SIZE*y+(x+1));
        *(grid+GRID_SIZE*y+x) = G * u1_1_0;
    }
    else if (x==0&&y==GRID_SIZE-1) 
    {// corner condition 2: x = 0, y = GRID_SIZE - 1
        float u1_0_y1 = *(grid_1+GRID_SIZE*(y-1)+x);
        *(grid+GRID_SIZE*y+x) = G * u1_0_y1;
    }
    else if (x==GRID_SIZE-1&&y==0) 
    {// corner condition 3: x = GRID_SIZE - 1, y = 0
        float u1_x1_0 = *(grid_1+GRID_SIZE*y+(x-1));
        *(grid+GRID_SIZE*y+x) = G * u1_x1_0;
    }
    else if (x==GRID_SIZE-1&&y==GRID_SIZE-1) 
    {
        // corner condition 4: x = y = GRID_SIZE - 1
        float u1_x_y1 = *(grid_1+GRID_SIZE*(y-1)+x);
        *(grid+GRID_SIZE*y+x) = G * u1_x_y1;
    }
    else
    {// center case: safe to compute
        float u1_x_y = *(grid_1+GRID_SIZE*y+x);
        float u2_x_y = *(grid_2+GRID_SIZE*y+x);

        
        float u1_1x_y = *(grid+GRID_SIZE*y+(x+1));
        float u1_x1_y = *(grid+GRID_SIZE*y+(x-1));

        float u1_x_1y = *(grid+GRID_SIZE*(y+1)+x);
        float u1_x_y1 = *(grid+GRID_SIZE*(y-1)+x);
                
        *(grid+GRID_SIZE*y+x) = (RHO*(u1_x1_y + u1_1x_y + u1_x_y1 + u1_x_1y - 4*u1_x_y) + 2*u1_x_y - (1-MIU)*u2_x_y )/(1+MIU);
    }
}

int print_grid(float *grid)
{
    printf("Size of gird: %d nodes\n", GRID_SIZE*GRID_SIZE);

    for(int y=0; y<GRID_SIZE; y++){
        for(int x=0; x<GRID_SIZE; x++){
            printf("(%d,%d): %f ", y, x, *(grid+GRID_SIZE*y+x));
        }
        printf("\n");
    }
    return 0;
}

int main(int argc, char* argv[])
{
    if(argc!=2)
    {
        printf("RuntimeError: Wrong inputs\n");
        printf("Correct Format: ./grid_%d_%d <number of iterations>\n", GRID_SIZE, GRID_SIZE);
        return 1;
    }

    int iter = atoi(argv[1]);

    float *grid = (float)calloc(GRID_SIZE*GRID_SIZE, sizeof(float));

    float *c_grid, *c_grid_1, *c_grid_2;
    hipMalloc((void **) &c_grid_1, GRID_SIZE*GRID_SIZE*sizeof(float));
    hipMemcpy(c_grid_1, grid, GRID_SIZE*GRID_SIZE*sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **) &c_grid_2, GRID_SIZE*GRID_SIZE*sizeof(float));
    hipMemcpy(c_grid_2, grid, GRID_SIZE*GRID_SIZE*sizeof(float), hipMemcpyHostToDevice);

    *(grid+GRID_SIZE*(GRID_SIZE/2-1)+(GRID_SIZE/2-1)) = 1.0f;

    hipMalloc((void **) &c_grid, GRID_SIZE*GRID_SIZE*sizeof(float));
    hipMemcpy(c_grid, grid, GRID_SIZE*GRID_SIZE*sizeof(float), hipMemcpyHostToDevice);

    for(int i = 0; i<iter; i++){
        simulation_kernel <<<1, GRID_SIZE*GRID_SIZE>>>(c_grid, c_grid_1, c_grid_2);
        hipDeviceSynchronize();

        hipMemcpy(c_grid_2, c_grid_1, GRID_SIZE*GRID_SIZE*sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(c_grid_1, c_grid, GRID_SIZE*GRID_SIZE*sizeof(float), hipMemcpyDeviceToDevice);
        hipMemcpy(grid, c_grid, GRID_SIZE*GRID_SIZE*sizeof(float), hipMemcpyDeviceToHost);

        printf("Iteration \#%d\n",i);
        print_grid(grid);

    }

    hipFree(c_grid);
    hipFree(c_grid_1);
    hipFree(c_grid_2);

    grid = NULL;
    free(grid);

}
