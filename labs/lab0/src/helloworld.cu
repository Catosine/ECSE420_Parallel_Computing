#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>


__global__ void kernel(void){
	printf("hello world from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main(void){
	kernel <<<10, 10>>> ();
	hipDeviceSynchronize();
	return 0;
}
