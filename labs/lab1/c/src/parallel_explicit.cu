#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "reader.h"
#include <stdlib.h>

__global__ void kernel(int* data, int* result, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size)
	{
		if (*(data+idx*3+2)==0)
		{
			//and
			*(result+idx) = *(data+idx*3) && *(data+idx*3+1);
		} 
		else if (*(data+idx*3+2)==1)
		{
			//or
			*(result+idx) = *(data+idx*3) || *(data+idx*3+1);
		}
		else if (*(data+idx*3+2)==2)
		{
			//nand
			*(result+idx) = !(*(data+idx*3) && *(data+idx*3+1));
		}
		else if (*(data+idx*3+2)==3)
		{
			//nor
			*(result+idx) = !(*(data+idx*3) || *(data+idx*3+1));
		}
		else if (*(data+idx*3+2)==4)
		{
			//xor
			*(result+idx) = *(data+idx*3) ^ *(data+idx*3+1);
		}
		else if (*(data+idx*3+2)==5)
		{
			//xnor
			*(result+idx) = !(*(data+idx*3) ^ *(data+idx*3+1));
		}
		else
		{
			//invalid
			*(result+idx) = -1;
		}
		//*(result+idx)=idx;
	}
}

int main(int argc, char *argv[])
{
	printf("ECSE 420 Lab 1: Logic Gates Simulation - parallel_explicit\n");
	if (argc != 4)
	{
		printf("RuntimeError: Wrong inputs.\n");
		printf("You should input follows this format: ./parallel_explicit <input_file_path> <input_file_length> <output_file_path>\n");
		return 1;
	}
	
	int size = atoi(argv[2]);
	int* file = (int *)calloc(size*3, sizeof(int));
	/*
	for(int i=0; i<size; i++)
	{
		*(file+i) = (int *)calloc(3, sizeof(int));
	}
	*/
	if(read_csv_array(argv[1], file, &size)==0){
		int *cuda_file;
		
		hipMalloc((void **) &cuda_file, size*3*sizeof(int));
		hipMemcpy(cuda_file, file, size*3*sizeof(int), hipMemcpyHostToDevice);

		int *cuda_output;
		hipMalloc((void **) &cuda_output, size*sizeof(int));
		
		int block = size/1024;
		if (size%1024) {
			block++;
		}

		kernel <<<block, 1024>>> (cuda_file, cuda_output, size);
		hipDeviceSynchronize();

		int *output = (int *)calloc(size, sizeof(int));
		hipMemcpy(output, cuda_output, size*sizeof(int), hipMemcpyDeviceToHost);
		
		save(argv[3], output, size);
		printf("Done\n");

		hipFree(cuda_file);
		hipFree(cuda_output);

		file = NULL;
		free(file);

		output = NULL;
		free(output);

		return 0;
	}
	else
	{
		return 1;
	}
	
}
