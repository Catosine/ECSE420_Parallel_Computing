#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "gputimer.h"
#define INPUT1_LEN 200001
#define INPUT2_LEN 50000
#define INPUT3_LEN 200000
#define INPUT4_LEN 10000
#define N_BLOCK 32
#define N_THREAD 10

__global__ void kernel(int* nodePtrs, int* nodeNeightbors, int* nodeStatus, int* currLevelNodes, int* idxCurrLevelNodes, int* outputQueue, int* idxOutputQueue)
{
    int idx = atomicAdd(idxCurrLevelNodes, 1);
    while(idx < INPUT4_LEN)
    {
        // get node
        int node = *(currLevelNodes+idx);
        int nbr_idx = *(nodePtrs+node);
        int nbr_end = *(nodePtrs+node+1);
        for(; nbr_idx<=nbr_end; nbr_idx++)
	    {   
            int nbr = *(nodeNeightbors+nbr_idx);
            // check if visited
            if(*(nodeStatus+nbr*4)==0)
            {
                atomicAdd(nodeStatus+nbr*4, 1);
                int gate = *(nodeStatus+nbr*4+1);
                int input1 = *(nodeStatus+nbr*4+2);
                int input2 = *(nodeStatus+node*4+3);
                int result = -3;
                //AND
                if (gate == 0) {result = (input1 & input2);}
                //OR
                else if (gate == 1) {result = (input1 | input2);}
                //NAND
                else if (gate == 2) {result = !(input1 & input2);}
                //NOR
                else if (gate == 3) {result = !(input1 | input2);}
                //XOR
                else if (gate == 4) {result = (input1 ^ input2);}
                //XNOR
                else if (gate == 5) {result = !(input1 ^ input2);}
                else {result = -1;}
                *(nodeStatus+nbr*4+3) = result;
                int oidx = atomicAdd(idxOutputQueue, 1);
                *(outputQueue+oidx) = nbr;
            }
        }
        // increment idx by 1
        idx = atomicAdd(idxCurrLevelNodes, 1);
    }
}

int readFile124(char* name, int* data)
{
    FILE* f = fopen(name, "r");
    int offset = 0;
    char* line = (char* )calloc(10, sizeof(char));
    if (f)
    {   
        // omit the first
        fgets(line, 10, f);
        while(fgets(line, 10, f))
        {
            *(data+offset) = atoi(line);
            offset++;
        }
    }
    free(line);
    line = NULL;
    fclose(f);
    return 0;
}

int readFile3(char* name, int* data)
{
    FILE* f = fopen(name, "r");
    int offset = 0;
    char* line = (char *)calloc(10, sizeof(char));
    if (f) 
    {
        // omit the first
        fgets(line, 10, f);
        while(fgets(line, 10, f))
        {
            *(data+offset+0) = *(line+0) - '0';
            *(data+offset+1) = *(line+2) - '0';
            *(data+offset+2) = *(line+4) - '0';
            if (*(line+6) == '-') 
            {
                *(data+offset+3) = -1;
            }
            else if (*(line+6) == '0') 
            {
                *(data+offset+3) = 0;
            }
            else if (*(line+6) == '1') 
            {
                *(data+offset+3) = 1;
            }
            offset+=4;
        }
    }
    free(line);
    line = NULL;
    fclose(f);
    return 0;
}

void sort(int *pointer, int size){
    //get from https://stackoverflow.com/questions/13012594/sorting-with-pointers-instead-of-indexes
    int *i, *j, temp;
    for(i = pointer; i < pointer + size; i++){
        for(j = i + 1; j < pointer + size; j++){
            if(*j < *i){
                temp = *j;
                *j = *i;
                *i = temp;
            }
        }
    }
}

void compareFiles(char *file_name1, char *file_name2)
{
//get from https://www.tutorialspoint.com/c-program-to-compare-two-files-and-report-mismatches
    FILE* fp1 = fopen(file_name1, "r");
    FILE* fp2 = fopen(file_name2, "r");
    // fetching character of two file
    // in two variable ch1 and ch2
    char ch1 = getc(fp1);
    char ch2 = getc(fp2);

    // error keeps track of number of errors
    // pos keeps track of position of errors
    // line keeps track of error line
    int error = 0, pos = 0, line = 1;

    // iterate loop till end of file
    while (feof(fp1) && feof(fp2))
    {
	printf("line: %d\n", line);
        printf("ch1: %c, ch2: %c\n", ch1, ch2);
	pos++;

        // if both variable encounters new
        // line then line variable is incremented
        // and pos variable is set to 0
        if (ch1 == '\n' && ch2 == '\n')
        {
            line++;
            pos = 0;
        }

        // if fetched data is not equal then
        // error is incremented
        if (ch1 != ch2)
        {
            error++;
            printf("Line Number : %d \tError"
                   " Position : %d \n", line, pos);
        }

        // fetching character until end of file
        ch1 = getc(fp1);
        ch2 = getc(fp2);
    }

    printf("Total Errors : %d\n", error);
}

void compareNextLevelNodeFiles(char *file_name1, char *file_name2)
{

    FILE* fp_1 = fopen(file_name1, "r");
    if (fp_1 == NULL){
        fprintf(stderr, "Couldn't open file for reading\n");
        exit(1);
    }

    FILE* fp_2 = fopen(file_name2, "r");
    if (fp_2 == NULL){
        fprintf(stderr, "Couldn't open file for reading\n");
        exit(1);
    }

    int counter = 0;
    int len_1;
    int len_2;
    int length_file_1 = fscanf(fp_1, "%d", &len_1);
    int length_file_2 = fscanf(fp_2, "%d", &len_2);

    if(length_file_1 != length_file_2){
        fprintf(stderr, "Wrong file length\n");
        exit(1);
    }
    int *input1 = (int *)malloc(len_1 * sizeof(int));
    int *input2 = (int *)malloc(len_2 * sizeof(int));

    int temp1;
    int temp2;

    while ((fscanf(fp_1, "%d", &temp1) == 1) && (fscanf(fp_2, "%d", &temp2) == 1)) {
        (input1)[counter] = temp1;
        (input2)[counter] = temp2;
        counter++;
    }

    sort(input1, len_1);
    sort(input2, len_2);

    for(int i=0; i< len_1; i++){
        if(input1[i] != input2[i]){
            fprintf(stderr, "Something goes wrong\n");
            exit(1);
        }
    }

    printf("No errors!\n");

}

int main(int argc, char* argv[])
{
    if (argc == 7)
    {
        // read input 1
        int* data1 = (int* )calloc(INPUT1_LEN, sizeof(int));
        readFile124(argv[1], data1);
        // read input 2
        int* data2 = (int* )calloc(INPUT2_LEN, sizeof(int));
        readFile124(argv[2], data2);
        // read input 3
        int* data3 = (int* )calloc(INPUT3_LEN*4, sizeof(int));
        readFile3(argv[3], data3);
        // read input 1
        int* data4 = (int* )calloc(INPUT4_LEN, sizeof(int));
        readFile124(argv[4], data4);
        // setup idxBfsQueue
        int idxOutputQueue = 0;
        // setup idxNextLevelNodes
        int idxCurrLevelNodes = 0;

        // cuda setup
        int *c_data1, *c_data2, *c_data3, *c_data4, *c_outputQueue, *c_idxCurrLevelNodes, *c_idxOutputQueue;
        hipMalloc((void **) &c_data1, INPUT1_LEN*sizeof(int));
        hipMalloc((void **) &c_data2, INPUT2_LEN*sizeof(int));
        hipMalloc((void **) &c_data3, INPUT3_LEN*4*sizeof(int));
        hipMalloc((void **) &c_data4, INPUT4_LEN*sizeof(int));
        hipMalloc((void **) &c_outputQueue, INPUT3_LEN*sizeof(int));
        hipMalloc((void **) &c_idxCurrLevelNodes, sizeof(int));
        hipMalloc((void **) &c_idxOutputQueue, sizeof(int));
        hipMemcpy(c_data1, data1, INPUT1_LEN*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(c_data2, data2, INPUT2_LEN*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(c_data3, data3, INPUT3_LEN*4*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(c_data4, data4, INPUT4_LEN*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(c_idxCurrLevelNodes, &idxCurrLevelNodes, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(c_idxOutputQueue, &idxOutputQueue, sizeof(int), hipMemcpyHostToDevice);

        // run
	GpuTimer timer;
	timer.Start();
        kernel <<<N_BLOCK, N_THREAD>>> (c_data1, c_data2, c_data3, c_data4, c_idxCurrLevelNodes, c_outputQueue, c_idxOutputQueue);
	    hipDeviceSynchronize();
	timer.Stop();
	printf("Global Queuing Kernel Runtime: %f ms\n", timer.Elapsed());
        // data3 retrival
        hipMemcpy(data3, c_data3, INPUT3_LEN*4*sizeof(int), hipMemcpyDeviceToHost);

        // outputs retrival
        hipMemcpy(&idxOutputQueue, c_idxOutputQueue, sizeof(int), hipMemcpyDeviceToHost);
        int* output = (int *)calloc(INPUT3_LEN, sizeof(int));
        hipMemcpy(output, c_outputQueue, INPUT3_LEN*sizeof(int), hipMemcpyDeviceToHost);

        // save gate status
        FILE *gateStatus = fopen(argv[5], "w");
        fprintf(gateStatus, "%d\n", INPUT3_LEN);
        for(int i=0; i<INPUT3_LEN; i++)
        {
            fprintf(gateStatus, "%d\n", *(data3+i*4+3));
        }
        fclose(gateStatus);

        // save output
        FILE *queueStatus = fopen(argv[6], "w");
        fprintf(queueStatus, "%d\n", idxOutputQueue);
        for(int i=0; i<idxOutputQueue; i++)
        {
            fprintf(queueStatus, "%d\n", *(output+i));
        }
        fclose(queueStatus);
	
	printf("Checking gq_nodeOutput.raw...\n");
        compareFiles(argv[5], "sol_nodeOutput.raw");
	printf("Checking gq_nextLevelNodes.raw...\n");
        compareNextLevelNodeFiles(argv[6], "sol_nextLevelNodes.raw");

        // clean up
        hipFree(c_data1);
        hipFree(c_data2);
        hipFree(c_data3);
        hipFree(c_data4);
        hipFree(c_outputQueue);
        hipFree(c_idxCurrLevelNodes);
        hipFree(c_idxOutputQueue);

        free(data1);
        free(data2);
        free(data3);
        free(data4);
        free(output);

        data1 = NULL;
        data2 = NULL;
        data3 = NULL;
        data4 = NULL;
        output = NULL;
    
    }
    else
    {
        printf("RuntimeError: Please follow the correct input format as: /global_queuing <path_to_input_1.raw> <path_to_input_2.raw> <path_to_input_3.raw> <path_to_input_4.raw> <output_nodeOutput_filepath> <output_nextLevelNodes_filepath> ");
        return 1;
    }
}
