#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "gputimer.h"
#define INPUT1_LEN 200001
#define INPUT2_LEN 50000
#define INPUT3_LEN 200000
#define INPUT4_LEN 10000

__global__ void kernel(int* nodePtrs, int* nodeNeightbors, int* nodeStatus, int* currLevelNodes, int* idxCurrLevelNodes, int* outputQueue, int sharedQueueSize)
{
    extern __shared__ int blockQueue[];
    __shared__ int blockCounter[1];
    __shared__ int outputBlockOffset[1];

    // initialize the counter
    if (threadIdx.x==0) 
    {
        *blockCounter = 0;
        *outputBlockOffset = 0;
    }

    __syncthreads();

    int idx = atomicAdd(idxCurrLevelNodes, 1);
    
    while(idx < INPUT4_LEN)
    {
        // get node
        int node = *(currLevelNodes+idx);
        int nbr_idx = *(nodePtrs+node);
        int nbr_end = *(nodePtrs+node+1);
        for(; nbr_idx<=nbr_end; nbr_idx++)
	    {   
            int nbr = *(nodeNeightbors+nbr_idx);
            
            atomicAdd(nodeStatus+nbr*4, 1);
            int gate = *(nodeStatus+nbr*4+1);
            int input1 = *(nodeStatus+nbr*4+2);
            int input2 = *(nodeStatus+node*4+3);
            int result = -3;
            //AND
            if (gate == 0) {result = (input1 & input2);}
            //OR
            else if (gate == 1) {result = (input1 | input2);}
            //NAND
            else if (gate == 2) {result = !(input1 & input2);}
            //NOR
            else if (gate == 3) {result = !(input1 | input2);}
            //XOR
            else if (gate == 4) {result = (input1 ^ input2);}
            //XNOR
            else if (gate == 5) {result = !(input1 ^ input2);}
            else {result = -1;}
            *(nodeStatus+nbr*4+3) = result;

            int bidx = atomicAdd(blockCounter, 1);
            if (bidx >= sharedQueueSize)
            {
                //copy to queue;
                __syncthreads();
                if (threadIdx.x==0)
                {
                    int boffset = atomicAdd(outputBlockOffset, 1);
                    memcpy(outputQueue+boffset*sharedQueueSize, blockQueue, sharedQueueSize*sizeof(int));
                    *blockCounter = 0;
                }
                __syncthreads();
            }
            //save to block mem
            bidx = atomicAdd(blockCounter, 1);
            *(blockQueue+bidx) = nbr;
            
        }
        // increment idx by 1
        idx = atomicAdd(idxCurrLevelNodes, 1);
    }

    __syncthreads();
    if (threadIdx.x==0)
    {
        int boffset = atomicAdd(outputBlockOffset, 0) - 1;
        memcpy(outputQueue+boffset*sharedQueueSize, blockQueue, atomicAdd(blockCounter, 1)*sizeof(int));            
    }
    __syncthreads();
}

int readFile124(char* name, int* data)
{
    FILE* f = fopen(name, "r");
    int offset = 0;
    char* line = (char* )calloc(10, sizeof(char));
    if (f)
    {   
        // omit the first
        fgets(line, 10, f);
        while(fgets(line, 10, f))
        {
            *(data+offset) = atoi(line);
            offset++;
        }
    }
    free(line);
    line = NULL;
    fclose(f);
    return 0;
}

int readFile3(char* name, int* data)
{
    FILE* f = fopen(name, "r");
    int offset = 0;
    char* line = (char *)calloc(10, sizeof(char));
    if (f) 
    {
        // omit the first
        fgets(line, 10, f);
        while(fgets(line, 10, f))
        {
            *(data+offset+0) = *(line+0) - '0';
            *(data+offset+1) = *(line+2) - '0';
            *(data+offset+2) = *(line+4) - '0';
            if (*(line+6) == '-') 
            {
                *(data+offset+3) = -1;
            }
            else if (*(line+6) == '0') 
            {
                *(data+offset+3) = 0;
            }
            else if (*(line+6) == '1') 
            {
                *(data+offset+3) = 1;
            }
            offset+=4;
        }
    }
    free(line);
    line = NULL;
    fclose(f);
    return 0;
}

void sort(int *pointer, int size){
    //get from https://stackoverflow.com/questions/13012594/sorting-with-pointers-instead-of-indexes
    int *i, *j, temp;
    for(i = pointer; i < pointer + size; i++){
        for(j = i + 1; j < pointer + size; j++){
            if(*j < *i){
                temp = *j;
                *j = *i;
                *i = temp;
            }
        }
    }
}

void compareFiles(char *file_name1, char *file_name2)
{
//get from https://www.tutorialspoint.com/c-program-to-compare-two-files-and-report-mismatches
    FILE* fp1 = fopen(file_name1, "r");
    FILE* fp2 = fopen(file_name2, "r");
    // fetching character of two file
    // in two variable ch1 and ch2
    char ch1 = getc(fp1);
    char ch2 = getc(fp2);

    // error keeps track of number of errors
    // pos keeps track of position of errors
    // line keeps track of error line
    int error = 0, pos = 0, line = 1;

    // iterate loop till end of file
    while (ch1 != EOF && ch2 != EOF)
    {
        pos++;

        // if both variable encounters new
        // line then line variable is incremented
        // and pos variable is set to 0
        if (ch1 == '\n' && ch2 == '\n')
        {
            line++;
            pos = 0;
        }

        // if fetched data is not equal then
        // error is incremented
        if (ch1 != ch2)
        {
            error++;
            printf("Line Number : %d \tError"
                   " Position : %d \n", line, pos);
        }

        // fetching character until end of file
        ch1 = getc(fp1);
        ch2 = getc(fp2);
    }

    printf("Total Errors : %d\t", error);
}

void compareNextLevelNodeFiles(char *file_name1, char *file_name2)
{

    FILE* fp_1 = fopen(file_name1, "r");
    if (fp_1 == NULL){
        fprintf(stderr, "Couldn't open file for reading\n");
        exit(1);
    }

    FILE* fp_2 = fopen(file_name2, "r");
    if (fp_2 == NULL){
        fprintf(stderr, "Couldn't open file for reading\n");
        exit(1);
    }

    int counter = 0;
    int len_1;
    int len_2;
    int length_file_1 = fscanf(fp_1, "%d", &len_1);
    int length_file_2 = fscanf(fp_2, "%d", &len_2);

    if(length_file_1 != length_file_2){
        fprintf(stderr, "Wrong file length\n");
        exit(1);
    }
    int *input1 = (int *)malloc(len_1 * sizeof(int));
    int *input2 = (int *)malloc(len_2 * sizeof(int));

    int temp1;
    int temp2;

    while ((fscanf(fp_1, "%d", &temp1) == 1) && (fscanf(fp_2, "%d", &temp2) == 1)) {
        (input1)[counter] = temp1;
        (input2)[counter] = temp2;
        counter++;
    }

    sort(input1, len_1);
    sort(input2, len_2);

    for(int i=0; i< len_1; i++){
        if(input1[i] != input2[i]){
            fprintf(stderr, "Something goes wrong\n");
            exit(1);
        }
    }

    fprintf(stderr, "No errors!\n");
    exit(1);

}

int main(int argc, char* argv[])
{
    if (argc == 10)
    {   
        // read parameters
        int numBlock = atoi(argv[1]);
        int blockSize = atoi(argv[2]);
        int sharedQueueSize = atoi(argv[3]);

        // read input 1
        int* data1 = (int* )calloc(INPUT1_LEN, sizeof(int));
        readFile124(argv[4], data1);
        // read input 2
        int* data2 = (int* )calloc(INPUT2_LEN, sizeof(int));
        readFile124(argv[5], data2);
        // read input 3
        int* data3 = (int* )calloc(INPUT3_LEN*4, sizeof(int));
        readFile3(argv[6], data3);
        // read input 1
        int* data4 = (int* )calloc(INPUT4_LEN, sizeof(int));
        readFile124(argv[7], data4);
        // setup idxNextLevelNodes
        int idxCurrLevelNodes = 0;

        // cuda setup
        int *c_data1, *c_data2, *c_data3, *c_data4, *c_outputQueue, *c_idxCurrLevelNodes;
        hipMalloc((void **) &c_data1, INPUT1_LEN*sizeof(int));
        hipMalloc((void **) &c_data2, INPUT2_LEN*sizeof(int));
        hipMalloc((void **) &c_data3, INPUT3_LEN*4*sizeof(int));
        hipMalloc((void **) &c_data4, INPUT4_LEN*sizeof(int));
        hipMalloc((void **) &c_outputQueue, INPUT3_LEN*sizeof(int));
        hipMalloc((void **) &c_idxCurrLevelNodes, sizeof(int));
        hipMemcpy(c_data1, data1, INPUT1_LEN*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(c_data2, data2, INPUT2_LEN*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(c_data3, data3, INPUT3_LEN*4*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(c_data4, data4, INPUT4_LEN*sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(c_idxCurrLevelNodes, &idxCurrLevelNodes, sizeof(int), hipMemcpyHostToDevice);

        // run
        GpuTimer timer;
        timer.Start();
        kernel <<<numBlock, blockSize, sharedQueueSize*sizeof(int)>>> (c_data1, c_data2, c_data3, c_data4, c_idxCurrLevelNodes, c_outputQueue, sharedQueueSize);
	    hipDeviceSynchronize();
        timer.Stop();
        printf("Block Queuing Kernel Runtime (blockSize=%d, numBlock=%d, blockQueueCapacity=%d): %f ms\n", numBlock, blockSize, sharedQueueSize, timer.Elapsed());

        // data3 retrival
        hipMemcpy(data3, c_data3, INPUT3_LEN*4*sizeof(int), hipMemcpyDeviceToHost);

        // outputs retrival
        int* output = (int *)calloc(INPUT3_LEN, sizeof(int));
        hipMemcpy(output, c_outputQueue, INPUT3_LEN*sizeof(int), hipMemcpyDeviceToHost);

        // save gate status
        FILE *gateStatus = fopen(argv[8], "w");
        fprintf(gateStatus, "%d\n", INPUT3_LEN);
        for(int i=0; i<INPUT3_LEN; i++)
        {
            fprintf(gateStatus, "%d\n", *(data3+i*4+3));
        }
        fclose(gateStatus);

        // save output
        FILE *queueStatus = fopen(argv[9], "w");
        fprintf(queueStatus, "%d\n", INPUT3_LEN);
        for(int i=0; i<INPUT3_LEN; i++)
        {
            fprintf(queueStatus, "%d\n", *(output+i));
        }
        fclose(queueStatus);

        compareFiles(argv[8], "sol_nodeOutput.raw");
        compareNextLevelNodeFiles(argv[9], "sol_nextLevelNodes.raw");

        // clean up
        hipFree(c_data1);
        hipFree(c_data2);
        hipFree(c_data3);
        hipFree(c_data4);
        hipFree(c_outputQueue);
        hipFree(c_idxCurrLevelNodes);

        free(data1);
        free(data2);
        free(data3);
        free(data4);
        free(output);

        data1 = NULL;
        data2 = NULL;
        data3 = NULL;
        data4 = NULL;
        output = NULL;
    
    }
    else
    {
        printf("RuntimeError: Please follow the correct input format as: /block_queuing <numBlock> <blockSize> <sharedQueueSize> <path_to_input_1.raw> <path_to_input_2.raw> <path_to_input_3.raw> <path_to_input_4.raw> <output_nodeOutput_filepath> <output_nextLevelNodes_filepath> ");
        return 1;
    }
}
